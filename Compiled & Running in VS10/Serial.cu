
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
//variabel global
int i,j,maks=0,minimal=9999999,summin,summaks;

int main(int argc, char *argv[]) {
    //Deklarasi Variabel
    FILE *baca; //untuk membuka file txt
    char buf[4]; //digunakan pada sesi membaca file txt
    int a[100][100],b; //matriks yang akan diisi oleh variabel b

    //Sesi 1: Membaca txt File
    baca=fopen("MatrixTubes1.txt","r");
    if (!baca){
        printf("File tidak ditemukan"); //Cek File ada atau tidak (perlukah?)
    }

    i=0;j=0; //inisialisi indeks matriks

    if(!feof(baca)){ //jika belum mencapai akhir file, maka
        for(i = 0; i < 100; i++){
            for(j = 0; j < 100; j++){
                fscanf(baca,"%s",buf); //baca file per maksimal 4 karakter
                b=atoi(buf); //parsing data string ke integer
                a[i][j]=b; //simpan data integer ke matriks
            }
        }
    }
    fclose(baca);
    //isi file telah dibaca dan seluruh matriks telah terisi
    //saatnya menutup file

    //Sesi 2: Cetak Seluruh Elemen Matriks
    for (i = 0; i < 100; i++){
        for (j = 0; j < 100; j++){
            printf("%d\t",a[i][j]);
        }
    }

    //Sesi 3: Pencarian Elemen Minimum dan Maksimum
    for (i = 0; i < 100; i++){
        for (j = 0; j < 100; j++){
            if(a[i][j]<minimal){
                minimal=a[i][j];
            }

            if(a[i][j]>maks){
                maks=a[i][j];
            }
        }
    }

    //inisialisasi nilai variabel
    summin=0; summaks=0;

    //Sesi 4: Mencari Frekuensi Kemunculan Angka Minimum & Maksimum
    for (i = 0; i < 100; i++){
        for (j = 0; j < 100; j++){
            if(a[i][j]==minimal){
                summin+=1;
            }

            if(a[i][j]==maks){
                summaks+=1;
            }
        }
    }

    //Sesi 5: Cetak Semua Hasil Pencarian
    printf("\n \n Angka Paling Kecil: %d \n Muncul sebanyak: %d kali \n \n Angka Paling Besar: %d \n Muncul sebanyak: %d kali \n",minimal,summin,maks,summaks);
    system("pause"); return 0;
}
