
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
//#include <sys/time.h>

//variabel global
int i,j,maks=0,minimal=9999999;

int main(int argc, char *argv[]) {
    //Deklarasi Variabel
    //struct timeval start, end; //Variabel execution timer
    //double delta1,delta2;
    FILE *baca; //untuk membuka file txt
    char buf[4]; //digunakan pada sesi membaca file txt
    int a[100][100],b; //matriks yang akan diisi oleh variabel b

	//Starting Clock Time 
	clock_t begin = clock();

    //Sesi 1: Membaca txt File
    baca=fopen("MatrixTubes1.txt","r");
    if (!baca){
        printf("File tidak ditemukan"); //Cek File ada atau tidak (perlukah?)
    }

    i=0;j=0; //inisialisi indeks matriks

    if(!feof(baca)){ //jika belum mencapai akhir file, maka
        for(i = 0; i < 100; i++){
            for(j = 0; j < 100; j++){
                fscanf(baca,"%s",buf); //baca file per maksimal 4 karakter
                b=atoi(buf); //parsing data string ke integer
                a[i][j]=b; //simpan data integer ke matriks
            }
        }
    }
    fclose(baca);
    //isi file telah dibaca dan seluruh matriks telah terisi
    //saatnya menutup file

    //Sesi 2: Cetak Seluruh Elemen Matriks
    for (i = 0; i < 100; i++){
        for (j = 0; j < 100; j++){
            printf("%d\t",a[i][j]);
        }
    }

    //Sesi 3: Pencarian Elemen Minimum dan Maksimum
    //gettimeofday(&start, NULL);
    for (i = 0; i < 100; i++){
        for (j = 0; j < 100; j++){

            if(a[i][j]<minimal){
                minimal=a[i][j];
            }

            if(a[i][j]>maks){
                maks=a[i][j];
            }
        }
    }
    //gettimeofday(&end, NULL);
    //delta1 = ((end.tv_sec  - start.tv_sec) * 1000000u + end.tv_usec - start.tv_usec) / 1.e6;

    //Sesi 4: Cetak Semua Hasil Pencarian
    printf("\n \n Angka Paling Kecil: %d \n Angka Paling Besar: %d \n",minimal,maks);
    //printf("\n CPU_time = %g \n",delta2);

	//Stop Clock Time
	clock_t end = clock();

	//Calculate Clock Time
	double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
	printf("\n Clock Time = %g \n",time_spent);
	
	system("pause"); return 0;
}


