
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

__global__ void FindMinMax(int *a, int *b, int *c) {   
	if(*a < *b){
	*b = *a;  
	}
	if(*a > *c){
	*c = *a;  
	}
}


int main(int argc, char *argv[]) {
//---------------------------------------------------------------------------------------------------------------------    
	//Deklarasi Variabel
    FILE *baca; //untuk membuka file txt
    char buf[4]; //digunakan pada sesi membaca file txt
	int a[100][100],b; //matriks yang akan diisi oleh variabel b
	int i,j,maks=0,min=9999; //variabel host lainnya
	int *d_a, *d_b, *d_c; //variabel device
	int size = sizeof(int); //ukuran data tiap variabel
//---------------------------------------------------------------------------------------------------------------------
	//Starting Clock Time 
	clock_t begin = clock();
//---------------------------------------------------------------------------------------------------------------------
    //Sesi 1: Alokasi ruang untuk variabel device
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size); 
//---------------------------------------------------------------------------------------------------------------------    
	
	//Sesi 2: Membaca txt File
    baca=fopen("MatrixTubes1.txt","r");
    if (!baca){
        printf("File tidak ditemukan"); //Cek File ada atau tidak (perlukah?)
    }

    i=0;j=0; //inisialisi indeks matriks

    if(!feof(baca)){ //jika belum mencapai akhir file, maka
        for(i = 0; i < 100; i++){
            for(j = 0; j < 100; j++){
                fscanf(baca,"%s",buf); //baca file per maksimal 4 karakter
                b=atoi(buf); //parsing data string ke integer
                a[i][j]=b; //simpan data integer ke matriks
            }
        }
    }
    fclose(baca);
    //isi file telah dibaca dan seluruh matriks telah terisi
    //saatnya menutup file
//---------------------------------------------------------------------------------------------------------------------
    //Sesi 2: Cetak Seluruh Elemen Matriks
    for (i = 0; i < 100; i++){
        for (j = 0; j < 100; j++){
            printf("%d\t",a[i][j]);
        }
    }
//---------------------------------------------------------------------------------------------------------------------
	for (i = 0; i < 100; i++){
        for (j = 0; j < 100; j++){
			//Copy data dari variabel host ke variabel device
			hipMemcpy(d_a, &a[i][j], size, hipMemcpyHostToDevice);   
			hipMemcpy(d_b, &min, size, hipMemcpyHostToDevice); 
			hipMemcpy(d_c, &maks, size, hipMemcpyHostToDevice);
			
			//Memanggil fungsi kernel
			FindMinMax<<<512,512>>>(d_a, d_b, d_c);

			//Copy data dari variabel device ke variabel host
			hipMemcpy(&min, d_b, size, hipMemcpyDeviceToHost);
			hipMemcpy(&maks, d_c, size, hipMemcpyDeviceToHost);
        }
    }
/*  cudaMemcpy(da, &a, size*sizeof(int), cudaMemcpyHostToDevice);
	cudaMemcpy(dmaks, &maks, sizeof(int), cudaMemcpyHostToDevice);
	cudaMemcpy(dmin, &minimal, sizeof(int), cudaMemcpyHostToDevice); */
//---------------------------------------------------------------------------------------------------------------------
/*	dim3 threadsPerBlock(100, 100);
    dim3 numBlocks(100 / threadsPerBlock.x, 100 / threadsPerBlock.y);
	FindMin<<<numBlocks, threadsPerBlock>>>(da,dmin);
	FindMin<<<numBlocks, threadsPerBlock>>>(da,dmaks); */
//---------------------------------------------------------------------------------------------------------------------
    //Sesi 7: Cetak data dari tiap variabel host yang sudah diolah oleh device
	printf("\n \n Angka Paling Kecil: %d \n \n Angka Paling Besar: %d \n\n",min,maks);
    //printf("\n Minimum Value = %d \n",min);
    //printf("\n Maximum Value = %d \n",maks);
//---------------------------------------------------------------------------------------------------------------------
    //Sesi 8: Bebaskan alokasi ruang dan akhiri program
	hipFree(d_a);hipFree(d_b);hipFree(d_c);
//---------------------------------------------------------------------------------------------------------------------
	//Stop Clock Time
	clock_t end = clock();

	//Calculate Clock Time
	double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
	printf("\n Clock Time = %g \n",time_spent);
//---------------------------------------------------------------------------------------------------------------------
	system("pause"); return 0;
}